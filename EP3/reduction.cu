#include "hip/hip_runtime.h"
// #include <stdlib.h>
extern "C" {
	#include "reduction.h"
}

__device__ int gpu_min(int x, int y){
	return y ^ ((x ^ y) & -(x < y));
}

// __device__ double randomSampleGPU(double min, double max)
// {
// 	double range = (max - min);
// 	double div = RAND_MAX / range;
// 	return min + (rand() / div);
// }

__global__ void create_f_array(double* f_array, double* f_squared_array, double* samples, int64_t n){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n){
		double sample = samples[id];
		f_array[id] = sample;
		f_squared_array[id] = sample; 
	}
}

__global__ void reductionGPU(double* f_array, double* f_squared_array, int64_t totalAmount){

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < (totalAmount/2)){
		f_array[id] = f_array[id] + f_array[id + (totalAmount/2)];
		f_squared_array[id] = f_squared_array[id] + f_squared_array[id + (totalAmount/2)];
	}

}

double integral(double a, double b, double sig)
{
	return (b - a) * (f + sig * sqrt((fsquared - pow(f, 2)) / N));
}


void reduceOnGPU(){
	int64_t totalAmount = nextPowerOfTwo(N);
	double *d_f_array;
	double *d_f_squared_array;
	double *d_samples;


	dim3 grid, block;
	block.x = 1024;
	grid.x = (totalAmount + block.x - 1) / block.x;

    hipSetDevice(1);

	hipMalloc((void**)&d_samples, N*sizeof(double));
	hipMalloc((void**)&d_f_array, totalAmount*sizeof(double));
	hipMalloc((void**)&d_f_squared_array, totalAmount*sizeof(double));

	hipMemset(d_f_array, 0, totalAmount*sizeof(double));
	hipMemset(d_f_squared_array, 0, totalAmount*sizeof(double));
	hipMemcpy(d_samples, samples, N*sizeof(double), hipMemcpyHostToDevice);


	create_f_array<<<grid, block>>>(d_f_array, d_f_squared_array, d_samples, N);

	while(totalAmount > 1){

		reductionGPU<<<grid, block>>>(d_f_array, d_f_squared_array, totalAmount);
		hipDeviceSynchronize();

		totalAmount /= 2;
		grid.x = (totalAmount + block.x - 1) / block.x;
	}
	hipMemcpy(&f, d_f_array, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&fsquared, d_f_array, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_f_array);
	hipFree(d_f_squared_array);

	printf("f = %lf\n", f);
	printf("f_squared = %lf\n", fsquared);

	printf("GPU\n");
	double integral_plus = 2 * integral(ZERO, 0.5, 1);
	double integral_minus = 2 * integral(ZERO, 0.5, -1);
	printf("Integrais: \nSoma: %lf\nSubracao: %lf\n", integral_plus, integral_minus);
	// printMatrix(output, MATRIX_WIDTH, MATRIX_WIDTH);

}