#include "hip/hip_runtime.h"
// #include <stdlib.h>
extern "C" {
	#include "GPUImplementation.h"
}

__device__ int gpu_min(int x, int y){
	return y ^ ((x ^ y) & -(x < y));
}

// __device__ double randomSampleGPU(double min, double max)
// {
// 	double range = (max - min);
// 	double div = RAND_MAX / range;
// 	return min + (rand() / div);
// }

__device__ double fun_gpu(double x, int64_t M, int64_t k)
{
	double result = (sin((2 * M + 1) * M_PI * x) * cos(2 * M_PI * k * x)) / sin(M_PI * x);
	return result;
}

__global__ void create_f_array(double* f_array, double* f_squared_array, double* samples, int64_t n, int64_t M, int64_t k){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n){
		double f_xi = fun_gpu(samples[id], M, k);
		f_array[id] = f_xi ;
		f_squared_array[id] = f_xi * f_xi ;
	}
}

__global__ void reductionGPU(double* f_array, double* f_squared_array, int64_t totalAmount){

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < (totalAmount/2)){
		f_array[id] = f_array[id] + f_array[id + (totalAmount/2)];
		f_squared_array[id] = f_squared_array[id] + f_squared_array[id + (totalAmount/2)];
	}

}


void calculate_f_GPU(){
	int64_t totalAmount = nextPowerOfTwo(N);
	double *d_f_array;
	double *d_f_squared_array;
	double *d_samples;

	struct timespec begin, end;

	dim3 grid, block;
	block.x = 1024;
	grid.x = (totalAmount + block.x - 1) / block.x;

    hipSetDevice(1);

	hipMalloc((void**)&d_samples, N*sizeof(double));
	hipMalloc((void**)&d_f_array, totalAmount*sizeof(double));
	hipMalloc((void**)&d_f_squared_array, totalAmount*sizeof(double));

	hipMemset(d_f_array, 0, totalAmount*sizeof(double));
	hipMemset(d_f_squared_array, 0, totalAmount*sizeof(double));

	printf("Time spent copying data");
	clock_gettime(CLOCK_REALTIME, &begin);
	hipMemcpy(d_samples, samples, N*sizeof(double), hipMemcpyHostToDevice);
	clock_gettime(CLOCK_REALTIME, &end);
	printTimeElapsed(begin, end);



	create_f_array<<<grid, block>>>(d_f_array, d_f_squared_array, d_samples, N, M, k);

	while(totalAmount > 1){

		reductionGPU<<<grid, block>>>(d_f_array, d_f_squared_array, totalAmount);
		hipDeviceSynchronize();

		totalAmount /= 2;
		grid.x = (totalAmount + block.x - 1) / block.x;
	}
	hipMemcpy(&f, d_f_array, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&fsquared, d_f_squared_array, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_f_array);
	hipFree(d_f_squared_array);

	f = f/N;
	fsquared = fsquared/N;

}