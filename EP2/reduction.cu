#include "hip/hip_runtime.h"
// #include <stdlib.h>
extern "C" {
	#include "reduction.h"
}

__global__ void reductionGPU(int* d_matrixList, int matrix_size, int totalAmount){

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < (totalAmount/2)*matrix_size){
		// printf("(%d, %d)\n",id, id + matrix_size*(totalAmount/2));
		// printf("%d, %d, : min (%d, %d) \n", id, id + matrix_size*(totalAmount/2),  d_matrixList[id], d_matrixList[id + matrix_size*(totalAmount/2)]);
		d_matrixList[id] = min(d_matrixList[id], d_matrixList[id + matrix_size*(totalAmount/2)]);
	}

}


void reduceOnGPU(int *matrixList, int matrixAmount){
	int totalAmount = nextPowerOfTwo(matrixAmount);
	int *d_matrixList;

	dim3 grid, block;
	block.x = 1024;
	grid.x = (totalAmount + block.x - 1) / block.x;

	int *output = (int*)malloc(MATRIX_SIZE * sizeof(int));

    hipSetDevice(0);


	hipMalloc( (void**)&d_matrixList, totalAmount*MATRIX_SIZE*sizeof(int) );

	hipMemcpy( d_matrixList, matrixList, totalAmount*MATRIX_SIZE*sizeof(int), hipMemcpyHostToDevice );
	while(totalAmount > 1){

		printf("totalAmount = %d\n", totalAmount);
		reductionGPU<<<grid, block>>>(d_matrixList, MATRIX_SIZE, totalAmount);
		hipDeviceSynchronize();
		// for (int l = 0; l < totalAmount; l++)
		// {
		// 	for (int i = 0; i < MATRIX_SIZE; i++)
		// 	{
		// 		matrixList[MATRIX_SIZE * l + i] = min(matrixList[MATRIX_SIZE * l + i], matrixList[MATRIX_SIZE * ((totalAmount/2) + l) + i]);
		// 	}
		// }
		totalAmount /= 2;
		grid.x = (totalAmount + block.x - 1) / block.x;
	}
	hipMemcpy( output, d_matrixList, MATRIX_SIZE*sizeof(int), hipMemcpyDeviceToHost );


	printMatrix(output, MATRIX_WIDTH, MATRIX_WIDTH);
}