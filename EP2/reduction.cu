#include "hip/hip_runtime.h"
// #include <stdlib.h>
extern "C" {
	#include "reduction.h"
}

__device__ int gpu_min(int x, int y){
	return y ^ ((x ^ y) & -(x < y));
}

__global__ void reductionGPU(int* d_matrixList, int matrix_size, int totalAmount){

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < (totalAmount/2)){
		d_matrixList[id] = gpu_min(d_matrixList[id], d_matrixList[id + (totalAmount/2)]);
	}

}


int* reduceOnGPU(int *matrixList, int matrixAmount){
	int totalAmount = nextPowerOfTwo(matrixAmount);
	int *d_matrixList;

	totalAmount = totalAmount*MATRIX_SIZE;

	dim3 grid, block;
	block.x = 1024;
	grid.x = (totalAmount + block.x - 1) / block.x;

	int *output = (int*)malloc(MATRIX_SIZE * sizeof(int));

    hipSetDevice(1);

	hipMalloc((void**)&d_matrixList, totalAmount*sizeof(int));

	hipMemcpy(d_matrixList, matrixList, totalAmount*sizeof(int), hipMemcpyHostToDevice);
	while(totalAmount > MATRIX_SIZE){

		reductionGPU<<<grid, block>>>(d_matrixList, MATRIX_SIZE, totalAmount);
		hipDeviceSynchronize();

		totalAmount /= 2;
		grid.x = (totalAmount + block.x - 1) / block.x;
	}
	hipMemcpy(output, d_matrixList, MATRIX_SIZE*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(matrixList);

	printMatrix(output, MATRIX_WIDTH, MATRIX_WIDTH);

	return output;
}