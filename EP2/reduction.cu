#include "hip/hip_runtime.h"
// #include <stdlib.h>
extern "C" { 
	#include "reduction.h"
}

__global__ void reductionGPU(int* d_matrixList, int matrix_size, int totalAmount){

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < totalAmount*matrix_size){
		printf("totalAmount = %d\n", totalAmount);
		// printf("(%d, %d)\n",id, id + matrix_size*(totalAmount/2));
		printf("%d, %d, : min (%d, %d) \n", id, id + matrix_size*(totalAmount/2),  d_matrixList[id], d_matrixList[id + matrix_size*(totalAmount/2)]);
		d_matrixList[id] = min(d_matrixList[id], d_matrixList[id + matrix_size*(totalAmount/2)]);
	}

}


void reduceOnGPU(int *matrixList, int matrixAmount){
	int totalAmount = nextPowerOfTwo(matrixAmount);
	int *d_matrixList;

	dim3 grid, block;
	block.x = 1024;
	grid.x = (totalAmount + block.x - 1) / block.x;

	int *output = (int*)malloc(9 * sizeof(int));

    hipSetDevice(0);


	hipMalloc( (void**)&d_matrixList, totalAmount*sizeof(int) );

	hipMemcpy( d_matrixList, matrixList, totalAmount*sizeof(int), hipMemcpyHostToDevice );
	while(totalAmount > 1){

		reductionGPU<<<grid, block>>>(d_matrixList, MATRIX_SIZE, totalAmount);
		hipDeviceSynchronize();
		// for (int l = 0; l < totalAmount; l++)
		// {
		// 	for (int i = 0; i < MATRIX_SIZE; i++)
		// 	{
		// 		matrixList[MATRIX_SIZE * l + i] = min(matrixList[MATRIX_SIZE * l + i], matrixList[MATRIX_SIZE * ((totalAmount/2) + l) + i]);
		// 	}
		// }
		totalAmount /= 2;
		grid.x = (totalAmount + block.x - 1) / block.x;
	}
	hipMemcpy( output, d_matrixList, MATRIX_SIZE*sizeof(int), hipMemcpyDeviceToHost );


	printMatrix(output, MATRIX_WIDTH, MATRIX_WIDTH);
}