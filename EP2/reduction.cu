#include "hip/hip_runtime.h"
// #include <stdlib.h>
extern "C" {
	#include "reduction.h"
}

__global__ void reductionGPU(int* d_matrixList, int matrix_size, int totalAmount){

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < (totalAmount/2)*matrix_size){
		d_matrixList[id] = min(d_matrixList[id], d_matrixList[id + matrix_size*(totalAmount/2)]);
	}

}


void reduceOnGPU(int *matrixList, int matrixAmount){
	int totalAmount = nextPowerOfTwo(matrixAmount);
	int *d_matrixList;

	dim3 grid, block;
	block.x = 1024;
	grid.x = (totalAmount + block.x - 1) / block.x;

	int *output = (int*)malloc(MATRIX_SIZE * sizeof(int));

    hipSetDevice(0);

	hipMalloc((void**)&d_matrixList, totalAmount*MATRIX_SIZE*sizeof(int));

	hipMemcpy(d_matrixList, matrixList, totalAmount*MATRIX_SIZE*sizeof(int), hipMemcpyHostToDevice);
	while(totalAmount > 1){
		
		reductionGPU<<<grid, block>>>(d_matrixList, MATRIX_SIZE, totalAmount);
		hipDeviceSynchronize();

		totalAmount /= 2;
		grid.x = (totalAmount + block.x - 1) / block.x;
	}
	hipMemcpy(output, d_matrixList, MATRIX_SIZE*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(matrixList);

	printMatrix(output, MATRIX_WIDTH, MATRIX_WIDTH);
}